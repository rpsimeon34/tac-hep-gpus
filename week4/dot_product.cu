
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// FIXME
	// Use atomicAdd	
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory 
	
	// Check memory allocation for errors

	// Copy the matrices on GPU
	
	// Check memory copy for errors

	// Define block/grid dimentions and launch kernel
	
	// Copy results back to host
	
    // Check copy for errors

	// Verify result

	// Free allocated memory
	
	return 0;

}