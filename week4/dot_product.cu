
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;
const int block_size = 32;
const int grid_size = DSIZE/block_size;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)

// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// FIXME
	// Use atomicAdd
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        atomicAdd(C,A[idx]*B[idx]);
    }
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory 
    hipMalloc(&d_A, DSIZE*sizeof(int));
    hipMalloc(&d_B, DSIZE*sizeof(int));
    hipMalloc(&d_C, sizeof(int));
	
	// Check memory allocation for errors
    cudaCheckErrors();

	// Copy the matrices on GPU
    hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(int), hipMemcpyHostToDevice);
	
	// Check memory copy for errors
    cudaCheckErrors();

	// Define block/grid dimentions and launch kernel
    dot_product<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);
    hipDeviceSynchronize();
    cudaCheckErrors();
	
	// Copy results back to host
	hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);

    // Check copy for errors
    cudaCheckErrors();

	// Verify result
    printf("Result is %d, expected %d\n",*h_C,DSIZE*a*b);

	// Free allocated memory
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
	
	return 0;

}
