#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;
const int block_size = 32;
const int grid_size = DSIZE/block_size;

// error checking macro
#define cudaCheckErrors()                                       \
	do {                                                        \
		hipError_t __err = hipGetLastError();                 \
		if (__err != hipSuccess) {                             \
			fprintf(stderr, "Error:  %s at %s:%d \n",           \
			hipGetErrorString(__err),__FILE__, __LINE__);      \
			fprintf(stderr, "*** FAILED - ABORTING***\n");      \
			exit(1);                                            \
		}                                                       \
	} while (0)

// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

	// FIXME
	// Use atomicAdd	
}


int main() {
	
	// Create the device and host pointers
	int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

	// Fill in the host pointers 
	h_A = new int[DSIZE];
	h_B = new int[DSIZE];
	h_C = new int;
	for (int i = 0; i < DSIZE; i++){
		h_A[i] = a;
		h_B[i] = b;
	}

	*h_C = 0;


	// Allocate device memory 
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*sizeof(float));
	
	// Check memory allocation for errors
    cudaCheckErrors("Error while allocating device memory");

	// Copy the matrices on GPU
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
	
	// Check memory copy for errors
    cudaCheckErrors("Error while copying vectors to device");

	// Define block/grid dimentions and launch kernel
    dot_product<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("Error while running kernel");
	
	// Copy results back to host
	
    // Check copy for errors

	// Verify result

	// Free allocated memory
	
	return 0;

}
