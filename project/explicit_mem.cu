#include "hip/hip_runtime.h"
// Version of stencil and matrix multiplication with explicit memory copies
// and default CUDA stream

#include <stdio.h>
#include <algorithm>

using namespace std;

#define N 512
#define RADIUS 3
#define BLOCK_SIZE 32
#define A_val 1
#define B_val 2

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)


__global__ void stencil_2d(int *in, int *out) {

    int gindex_x = threadIdx.x + blockIdx.x * blockDim.x;
    int gindex_y = threadIdx.y + blockIdx.y * blockDim.y;

    int size = N + 2*RADIUS;

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++){
        result += in[gindex_y+(gindex_x+offset)*size];
        result += in[gindex_y+offset+gindex_x*size];
    }
    // Avoid double-counting the center
    result -= in[gindex_y+gindex_x*size];

    //FIXME
    // Store the result
    out[gindex_y+size*gindex_x] = result;
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    // create thread x index
    // create thread y index
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        int temp = 0;
        for (int i = 0; i < size; i++){
            temp += A[idy*size+i]*B[i*size+idx];
        }
        C[idy*size+idx] = temp;                    
    }

}


void fill_ints(int *x, int n, int val) {
   // Store the result
   // https://en.cppreference.com/w/cpp/algorithm/fill_n
   fill_n(x, n, val);
}


int main(void) {

    int *in, *out; // host copies of a, b, c
    int *d_in, *d_out; // device copies of a, b, c

    // Alloc space for host copies and setup values
    int size = (N + 2*RADIUS)*(N + 2*RADIUS) * sizeof(int);
    int DSIZE = N + 2*RADIUS;
    in_A = (int *)malloc(size); fill_ints(in, (N + 2*RADIUS)*(N + 2*RADIUS),A_val);
    out_A = (int *)malloc(size); fill_ints(out, (N + 2*RADIUS)*(N + 2*RADIUS),A_val);
    in_B = (int *)malloc(size); fill_ints(in, (N + 2*RADIUS)*(N + 2*RADIUS),B_val);
    out_B = (int *)malloc(size); fill_ints(out, (N + 2*RADIUS)*(N + 2*RADIUS),B_val);
    h_C = (int *)malloc(size); fill_ints(out, (N + 2*RADIUS)*(N + 2*RADIUS),0);

    // Alloc space for device copies
    hipMalloc((void **)&d_in_A, size);
    hipMalloc((void **)&d_out_A, size);
    hipMalloc((void **)&d_in_B, size);
    hipMalloc((void **)&d_out_B, size);
    hipMalloc((void **)&d_C, size);
    cudaCheckErrors("Error while allocating device memory");

    // Copy to device
    hipMemcpy(d_in_A, in_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_out_A, out_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_in_B, in_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_out_B, out_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    cudaCheckErrors("Error while copying from host to device");

    // Launch stencil_2d() kernel on GPU
    int gridSize = (N + BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 grid(gridSize, gridSize);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    // Launch the kernel 
    // Properly set memory address for first element on which the stencil will be applied
    stencil_2d<<<grid,block>>>(d_in_A + RADIUS*(N + 2*RADIUS) + RADIUS , d_out_A + RADIUS*(N + 2*RADIUS) + RADIUS);
    stencil_2d<<<grid,block>>>(d_in_B + RADIUS*(N + 2*RADIUS) + RADIUS , d_out_B + RADIUS*(N + 2*RADIUS) + RADIUS);
    cudaCheckErrors("Error while launching stencil kernel");

    // Launch mat_mult kernel on GPU
    int m_gridSize = (DSIZE + BLOCK_SIZE-1)/BLOCK_SIZE;
    dim3 m_grid(m_gridSize, m_gridSize);
    dim3 m_block(BLOCK_SIZE, BLOCK_SIZE);
    // Launch the kernel
    matrix_mul_gpu<<<m_grid, m_block>>>(d_out_A, d_out_B, d_C, DSIZE);
    cudaCheckErrors("Error while launching multiplication kernel");

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    cudaCheckErrors("Error while copying from device to host");

    // Error Checking
    int exp_edge = A_val*B_val*((RADIUS*4+1)*(DSIZE-2*RADIUS)+2*RADIUS));
    for (int i = 0; i < N + 2 * RADIUS; ++i) {
        for (int j = 0; j < N + 2 * RADIUS; ++j) {

            if ((i < RADIUS || i >= N + RADIUS) && (j < RADIUS || i >= N+RADIUS)) {
                if (out[j+i*(N + 2 * RADIUS)] != A_val*B_val*DSIZE) {
                    printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], A_val*B_val*DSIZE);
                    return -1;
                }
            }
            else if ((j < RADIUS || j >= N + RADIUS) && (i >= RADIUS && i< N+RADIUS)){
                if (out[j+i*(N + 2 * RADIUS)] != exp_edge) {
                    printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], exp_edge);
                    return -1;
                }
            }        
            else if ((i < RADIUS || i >= N + RADIUS) && (j >= RADIUS && j< N+RADIUS)){
                if (out[j+i*(N + 2 * RADIUS)] != exp_edge) {
                    printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], exp_edge);
                    return -1;
                }
            }
            else {
                if (out[j+i*(N + 2 * RADIUS)] != exp_edge*(RADIUS*4+1)) {
                    printf("Mismatch at index [%d,%d], was: %d, should be: %d\n", i,j, out[j+i*(N + 2 * RADIUS)], exp_edge*(1+4*RADIUS));
                    return -1;
                }
            }
        }
    }

    // Cleanup
    free(in_A);
    free(out_A);
    free(in_B);
    free(out_B);
    free(h_C);
    hipFree(d_in_A);
    hipFree(d_out_A);
    hipFree(d_in_B);
    hipFree(d_out_B);
    hipFree(d_C);
    printf("Success!\n");

    return 0;
}
