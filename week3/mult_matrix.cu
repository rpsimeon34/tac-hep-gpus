
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
    // assume matrices are NxN
    float sum;
    for (int i=0;i<size;i++) {
        for (int j=0;j<size;j++) {
            sum = 0;
            for (int k=0;k<size;k++) {
                sum += A[i*size+k]*B[k*size+j];
            }
            C[i*size+j] = sum;
        }
    }
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    // create thread x index
    // create thread y index
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            temp += A[idy*size+i]*B[i*size+idx];
        }
        C[idy*size+idx] = temp;                    
    }

}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3, t4, t5;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;
    double t4sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    //FIXME:Add all other allocations and copies from host to device
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
    cudaCheckErrors("Error while allocating device memory");

    hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Error while copying from host to device");
  

    // Launch kernel
    // Specify the block and grid dimentions 
    dim3 block(32,32);  //FIXME
    dim3 grid((DSIZE+31)/32,(DSIZE+31/32)); //FIXME
    t4 = clock();
    matrix_mul_gpu<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("Error while launching the kernel");
    t5 = clock();
    t4sum = ((double)(t5-t4))/CLOCKS_PER_SEC;
    printf("Executing kernel took %f seconds\n", t4sum);

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Error while copying from device to host");

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. GPU compute took %f seconds\n", t2sum);
    printf("%f seconds of GPU compute due to data transfer\n", t2sum-t4sum);

    // FIXME
    // Excecute and time the cpu matrix multiplication function
    matrix_mul_cpu(h_A, h_B, h_C, DSIZE);

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. CPU compute took %f seconds\n", t3sum);

    printf("C[0,0] is %f, expected %f\n",h_C[0],DSIZE*A_val*B_val);

    // FIXME
    // Free memory 
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;

}
