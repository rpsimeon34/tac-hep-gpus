#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

const int DSIZE = 256;
const float A_val = 3.0f;
const float B_val = 2.0f;

// error checking macro
#define cudaCheckErrors(msg)                                   \
   do {                                                        \
       hipError_t __err = hipGetLastError();                 \
       if (__err != hipSuccess) {                             \
           fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                   msg, hipGetErrorString(__err),             \
                   __FILE__, __LINE__);                        \
           fprintf(stderr, "*** FAILED - ABORTING\n");         \
           exit(1);                                            \
       }                                                       \
   } while (0)

// Square matrix multiplication on CPU : C = A * B
void matrix_mul_cpu(const float *A, const float *B, float *C, int size) {
  //FIXME:
  // 
}

// Square matrix multiplication on GPU : C = A * B
__global__ void matrix_mul_gpu(const float *A, const float *B, float *C, int size) {

    //FIXME:
    // create thread x index
    // create thread y index
    idx = ;
    idy = ;
    // Make sure we are not out of range
    if ((idx < FIXME) && (idy < FIXME)) {
        float temp = 0;
        for (int i = 0; i < size; i++){
            //FIXME : Add dot product of row and column
        }
        C[idy*size+idx] = temp;                    
    }

}

int main() {

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // These are used for timing
    clock_t t0, t1, t2, t3;
    double t1sum=0.0;
    double t2sum=0.0;
    double t3sum=0.0;

    // start timing
    t0 = clock();

    // N*N matrices defined in 1 dimention
    // If you prefer to do this in 2-dimentions cupdate accordingly
    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data from host to device
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    //FIXME:Add all other allocations and copies from host to device
  

    // Launch kernel
    // Specify the block and grid dimentions 
    dim3 block(,);  //FIXME
    dim3 grid(,); //FIXME
    matrix_mul_gpu<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    // FIXME
    // Excecute and time the cpu matrix multiplication function

    // CPU timing
    t3 = clock();
    t3sum = ((double)(t3-t2))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t3sum);

    // FIXME
    // Free memory 
    
    return 0;

}
